
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCK_SIZE 256
#define N 1000

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void prefix(int * X, int * Y) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i < N) {
     int sum = 0;
     for (int j = 0; j <= i; j++) {
         sum += X[j];
     }
     Y[i] = sum;
  }
}

int main() {
  int *x, *d_x, *y, *d_y;
  x = (int*)malloc(N*sizeof(int));
  y = (int*)malloc(N*sizeof(int));
  hipMalloc(&d_x, N*sizeof(int));
  hipMalloc(&d_y, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    x[i] = 1;
  }

  hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);

  double t0 = get_clock();
  prefix<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_x,d_y);
  hipDeviceSynchronize();
  double t1 = get_clock();

  hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

  printf("%s\n", hipGetErrorString(hipGetLastError()));
  for (int j = 0; j < N; j++) {
    printf("%d ", y[j]);
  }
  printf("\n");
  printf("Time: %f ns\n", (1000000000.0*(t1-t0)));

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}