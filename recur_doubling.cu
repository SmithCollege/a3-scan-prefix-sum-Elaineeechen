
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCK_SIZE 256
#define N 1000

double get_clock() {
  struct timeval tv; int ok;
  ok = gettimeofday(&tv, (void *) 0);
  if (ok<0) { printf("gettimeofday error"); }
  return (tv.tv_sec * 1.0 + tv.tv_usec * 1.0E-6);
}

__global__ void prefix(int * X, int * Y, int offset) {
  int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i < N) {
     if (i < offset) {
        Y[i] = X[i];
     } else {
       Y[i] = X[i] + X[i - offset];
     }
  }
}

int main(void) {
  int *x, *d_x, *y, *d_y;
  x = (int*)malloc(N*sizeof(int));
  y = (int*)malloc(N*sizeof(int));
  hipMalloc(&d_x, N*sizeof(int));
  hipMalloc(&d_y, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    x[i] = 1;
  }
  hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);

  double t0 = get_clock();
  for (int offset = 1; offset < N; offset *= 2) {
        prefix<<<(N+BLOCK_SIZE-1)/BLOCK_SIZE, BLOCK_SIZE>>>(d_x,d_y,offset);
        int* temp = d_y;
        d_y = d_x;
        d_x = temp;
  }
  hipDeviceSynchronize();
  double t1 = get_clock();

  hipMemcpy(y, d_x, N*sizeof(int), hipMemcpyDeviceToHost);

  printf("%s\n", hipGetErrorString(hipGetLastError()));
  for (int j = 0; j < N; j++) {
    printf("%d ", y[j]);
  }
  printf("\n");

  printf("Time: %f ns\n", (1000000000.0*(t1-t0)));

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);

  return 0;
}